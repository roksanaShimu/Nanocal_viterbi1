#include "hip/hip_runtime.h"


#include<iostream>
#include<vector>
#include<stdlib.h>
#include <utility>

#include "Initialization.hpp"
#include "Predefine_Values.hpp"
#include "Data_Structure.hpp"
#include "Serial_Implementation.hpp"
#include "Parallel_Implementation_1i.hpp"
//#include "Parallel_Implementation_1ii.hpp"
//#include "Parallel_Implementation_1ib.hpp"

using namespace std;


int main(){
	unsigned n_states=4096*2;
	unsigned n_events=2972*1;
	vector<Matrix_Entry> m; m.resize(n_states*n_events);  // need to calculate the values of m
	//fill m for first event;
	fill_randomly_m_for_first_event(m, n_states, n_events);

	//for error checking 
	vector<Matrix_Entry> m1; m1.resize(n_states*n_events);m1=m;
	vector<Matrix_Entry> m2; m2.resize(n_states*n_events);m2=m;

	vector< vector<pair<unsigned, float> > > neighbors_from_v; neighbors_from_v.resize(n_states);
	for(unsigned i=0; i<n_states; i++){
		neighbors_from_v[i].resize(length_of_from_v);
	}
	generate_random_values_for_neighbors_from_v(neighbors_from_v);
	//print_neighbors_from_v(neighbors_from_v);
	

	vector<pore_model_state> pm; pm.resize(n_states);
	generate_random_values_for_pm(pm, n_states);

	
	vector<event> ev; ev.resize(n_events);
	generate_random_values_for_event(ev);
	//print_event(ev);
	

	clock_t serial_start = clock();
	serial_calculations(n_states, m1, neighbors_from_v, ev, pm);
	clock_t serial_end = clock();

	cout<< "serial calculation is over"<<endl;
	printf("Time taken for serial_code: %.6fs\n", (double)(serial_end - serial_start)/CLOCKS_PER_SEC);

	clock_t parallel_start = clock();
	parallel_calculations_1(n_states, m2, neighbors_from_v, ev, pm);
	clock_t parallel_end = clock();


	clock_t parallel_1ii_start = clock();
	//parallel_calculations_1ii(n_states, m2, neighbors_from_v, ev, pm);
	clock_t parallel_1ii_end = clock();

	clock_t parallel_1ib_start = clock();
	//parallel_calculations_1ib(n_states, m2, neighbors_from_v, ev, pm);
	clock_t parallel_1ib_end = clock();


	printf("Time taken for serial_code: %.6fs\n", (double)(serial_end - serial_start)/CLOCKS_PER_SEC);
	printf("Time taken for parallel: %.6fs\n", (double)(parallel_end - parallel_start)/CLOCKS_PER_SEC);
	printf("Time taken for parallel: %.6fs\n", (double)(parallel_1ii_end - parallel_1ii_start)/CLOCKS_PER_SEC);
	printf("Time taken for parallel: %.6fs\n", (double)(parallel_1ib_end - parallel_1ib_start)/CLOCKS_PER_SEC);



 
	int error=0;
	int counter=0;
	for(unsigned i=0; i<m.size(); i++){
		if(m1[i].beta != m2[i].beta){
			//cout<<i<<"th value didn't matched. m1[i].beta= "<<m1[i].beta<<",  m1[i].alpha= "<< m1[i].alpha<< "  and m2[i].beta= "<< m2[i].beta<<", m2[i].alpha= "<<m2[i].alpha<<endl;
			error=1;
			counter++;
			//break;
		}
	}
	if(error==0){
		cout<<"all matched"<<endl;
	}else{
		cout<<counter<<" elements didn't matched"<<endl;
	}
	return 0;
}

